#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include "kmeans.h"
#include "kmeansGPU.h"
#include "error.cuh"

// 读取合成数据集，并将所有数据放到一个一维数组中
// data: 指向浮点数数组的指针，用于存储读取的特征数据
// label: 指向整型数组的指针，用于存储读取的标签数据
// n_features: 每个样本的特征数量
// n: 引用参数，用于存储读取的样本数量
void readCoordinate(float* data, int* label, const int n_features, int& n) {
    std::ifstream ifs;
    ifs.open("./synthetic_dataset.csv", std::ios::in);
    if (ifs.fail()) {
        std::cout << "No such file or directory: synthetic_dataset.csv"
            << std::endl;
        exit(1);
    }
    std::string line;
    while (std::getline(ifs, line)) {
        std::stringstream sstream(line);
        if (line.empty())
            continue;
        int m = 0;  // 跟踪当前行中已解析的字段数量
        std::string s_fea;
        while (std::getline(sstream, s_fea, ',')) {
            if (m < n_features)
                data[n * n_features + m] =
                std::stod(s_fea);  // 将前 n_features
            // 个字段（特征值）转换为浮点数，并存储到
            // data 数组中。
            else
                label[n] =
                std::stoi(s_fea);  // 将第 n_features
            // 个字段（标签）转换为整数，并存储到
            // label 数组中。
            m++;
        }
        n++;
    }
    ifs.close();
}

// timing 函数用于测量 KMeans 聚类算法的运行时间
void timing(
    float* data,            // 指向浮点数数组的指针，用于存储读取的特征数据
    int* label,             // 指向整型数组的指针，用于存储读取的标签数据
    float* clusters,        // 存储初始聚类中心的数组
    const int numClusters,  // 聚类中心数量
    const int n_features,   // 特征数量
    const int n_samples,    // 样本数量
    const int method        // 方法选择，0表示CPU，1表示GPU
) {

    Kmeans* model;

    switch (method) {
    case 0: // CPU
        model = new Kmeans(numClusters, n_features, clusters, n_samples, 50, 0.1);
        break;
    case 1: // GPU
        model = new KmeansGPU(numClusters, n_features, clusters, n_samples, 50, 0.1);
        break;
    default:
        std::cout << "method not supported!" << std::endl;
        break;
    }

    std::cout << "*********starting fitting*********" << std::endl;

    hipEvent_t start, stop;    // CUDA事件，用于测量时间
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);
    model->fit(data);

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    printf("Time = %g ms.\n", elapsedTime);

    std::cout << "********* final clusters**********" << std::endl;
    std::cout << "*********    accuracy  **********" << std::endl;
    std::cout << "model accuracy : " << model->accuracy(label) << std::endl;

    std::cout << "*********    result saving   **********" << std::endl;
    model->saveLabels();
    std::cout << "*********    result saving done   **********" << std::endl;
    delete model;
}

int main(int argc, char* argv[]) {
    int N = 0;  // 样本数量
    int n_features = 100;   // 特征数量
    const int bufferSize = 10000 * n_features;  // 缓冲区大小
    float* data = new float[bufferSize];    // 指向浮点数数组的指针，用于存储读取的特征数据
    int* label = new int[bufferSize];       // 指向整型数组的指针，用于存储读取的标签数据
    readCoordinate(data, label, n_features, N); // 读取数据集
    std::cout << "num of samples : " << N << std::endl;
    int cidx[] = { 1, 3, 6, 8 };    // 选择的初始聚类中心的索引
    int numClusters = 4;    // 聚类中心数量
    float clusters[400] = { 0 };    // 存储初始聚类中心的数组
    for (int i = 0; i < numClusters; ++i) { // 将初始聚类中心的坐标从数据集中复制到 clusters 数组中
        for (int j = 0; j < n_features; ++j) {
            clusters[i * n_features + j] = data[cidx[i] * n_features + j];
        }
    }
    std::cout << "********* init clusters **********" << std::endl;
    std::cout << "Using CPU:" << std::endl;
    timing(data, label, clusters, numClusters, n_features, N, 0);
    std::cout << "Using CUDA:" << std::endl;
    timing(data, label, clusters, numClusters, n_features, N, 1);

    delete[] data;
    delete[] label;

    return 0;
}